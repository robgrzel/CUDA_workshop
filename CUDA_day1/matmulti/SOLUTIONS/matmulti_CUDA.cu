/* using column major storage 

Compile on monk with:

nvcc -arch=sm_20 -O2 matmulti_CUDA.cu -o matmulti_CUDA.x  

Program must be run on system with a working CUDA GPU 






*/


#include <hip/hip_runtime.h> /* CUDA runtime API */
#include <cstdio> 
#include <math.h>
#include <sys/time.h>
#include <time.h>
#include <hipblas.h>

#define TILE_DIM 32 

int
timeval_subtract (double *result, struct timeval *x, struct timeval *y)
{
  struct timeval result0;

  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result0.tv_sec = x->tv_sec - y->tv_sec;
  result0.tv_usec = x->tv_usec - y->tv_usec;
  *result = ((double)result0.tv_usec)/1e6 + (double)result0.tv_sec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}

int check_float(float x,float y)
{
float rel_err; 
float tol=0.00001f;

rel_err= fabsf(x-y) / y;

if(rel_err>tol) return 0;
else return 1;

}

void simpleMultiply_cpu(float *a, float *b, float *c ,int N)
{
   float sum;

   for (int col=0; col < N ; col++ ){
     for (int row=0; row < N ; row++ ){

       sum = 0.0f;
       for (int i=0; i < TILE_DIM; i++) {
          sum+= a[row+i*N]*b[i+col*TILE_DIM];
       }
       c[row+col*N]=sum;
     }
   
   }
}

__global__ void simpleMultiply_gpu(float *a, float *b, float *c ,int N)
{

  int col=blockIdx.y*blockDim.y + threadIdx.y;
  int row=blockIdx.x*blockDim.x + threadIdx.x;

  float sum = 0.0f;

  for (int i=0; i < TILE_DIM; i++) {
      sum+= a[row+i*N]*b[i+col*TILE_DIM];
  }
  c[row+col*N]=sum;
}

__global__ void sharedmemoryMultiply_gpu(float *a, float *b, float *c ,int N)
{
  __shared__ float aTile[TILE_DIM*TILE_DIM], bTile[TILE_DIM*TILE_DIM];

  int col=blockIdx.y*blockDim.y + threadIdx.y;
  int row=blockIdx.x*blockDim.x + threadIdx.x;

  float sum = 0.0f;

  aTile[threadIdx.y*TILE_DIM+threadIdx.x] = a[row+threadIdx.y*N];
  bTile[threadIdx.y*TILE_DIM+threadIdx.x] = b[threadIdx.x+col*TILE_DIM];
  __syncthreads();
  for (int i=0; i < TILE_DIM; i++) {
      sum+= aTile[threadIdx.x+ i*TILE_DIM]*bTile[i+threadIdx.y*TILE_DIM];
  }
  c[row+col*N]=sum;
}


int main(int argc, char *argv[])
{
   float *a_host, *b_host, *c_host;   /* arrays for computation on host*/
   float *a_dev, *b_dev, *c_dev;     /* arrays for computation on device */
   float *c_shadow;          /* host-side copy of device results */

   int n = 512 * TILE_DIM ;
   int nerror;
   double restime;

   size_t memsize_input,memsize_output;

   /* set up device execution configuration */
   dim3 gridDef(n/TILE_DIM,n/TILE_DIM);
   dim3 blockDef(TILE_DIM,TILE_DIM,1);

   struct timeval  tdr0, tdr1;
   int error;
   hipEvent_t start, stop;

   /*  find compute device an initialize it */
   /* add device detection */

   memsize_input = n * TILE_DIM * sizeof(float);
   memsize_output = n * n * sizeof(float);
   /* allocate arrays on host */

   if(error = hipHostMalloc((void **) &a_host, memsize_input, hipHostMallocDefault))
{
      printf ("Error in hipHostMalloc %d\n", error);
      exit (error);
}

   if(error = hipHostMalloc((void **) &b_host, memsize_input, hipHostMallocDefault))
{
      printf ("Error in hipHostMalloc %d\n", error);
      exit (error);
}

   if(error = hipHostMalloc((void **) &c_host, memsize_output, hipHostMallocDefault))
{
      printf ("Error in hipHostMalloc %d\n", error);
      exit (error);
}

   if(error = hipHostMalloc((void **) &c_shadow, memsize_output, hipHostMallocDefault))
{
      printf ("Error in hipHostMalloc %d\n", error);
      exit (error);
}



   /* allocate arrays on device */

   if(error = hipMalloc((void **) &a_dev, memsize_input))
    {
      printf ("Error in hipMalloc %d\n", error);
      exit (error);
    }

   if(error = hipMalloc((void **) &b_dev, memsize_input))
    {
      printf ("Error in hipMalloc %d\n", error);
      exit (error);
    }

   if(error = hipMalloc((void **) &c_dev, memsize_output))
    {
      printf ("Error in hipMalloc %d\n", error);
      exit (error);
    }

   /* initialize arrays on host */

   for ( int i = 0; i < n*TILE_DIM; i++)
   {
      a_host[i] = rand() / (float)RAND_MAX;
      b_host[i] = rand() / (float)RAND_MAX;
   }

   /* copy arrays to device memory (synchronous) */

  gettimeofday (&tdr0, NULL);

  if (error = hipMemcpy(a_dev, a_host, memsize_input, hipMemcpyHostToDevice))
    {
      printf ("Error %d\n", error);
      exit (error);
    }

  if (error = hipMemcpy(b_dev, b_host, memsize_input, hipMemcpyHostToDevice))
    {
      printf ("Error %d\n", error);
      exit (error);
    }

  float kernel_timer;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  simpleMultiply_gpu<<<gridDef,blockDef>>>(a_dev, b_dev, c_dev, n);

  hipEventRecord(stop, 0);
  hipEventSynchronize( stop ); 
  hipEventElapsedTime( &kernel_timer, start, stop );

  printf("Test Kernel took %f ms\n",kernel_timer);
  printf ("GFlops %f\n",  (float)(n*n)*(2.0f*(float)TILE_DIM-1.0f)/kernel_timer/1000000.0f);

   /* retrieve results from device (synchronous) */
  if (error =  hipMemcpy(c_shadow, c_dev, memsize_output, hipMemcpyDeviceToHost))
    {
      printf ("Error %d\n", error);
      exit (error);
    }

  gettimeofday (&tdr1, NULL);
  timeval_subtract (&restime, &tdr1, &tdr0);
  printf ("gpu kernel and memcopy%e\n", restime);
  printf ("GFlops %e\n",  (float)(n*n)*(2.0f*(float)TILE_DIM-1.0f)/restime/1000000000.0f);

  gettimeofday (&tdr0, NULL);
   /* execute host version (i.e. baseline reference results) */
   simpleMultiply_cpu(a_host, b_host, c_host, n);

  gettimeofday (&tdr1, NULL);
  timeval_subtract (&restime, &tdr1, &tdr0);
  printf ("cpu kernel %e\n", restime);
  printf ("GFlops %e\n",  (float)(n*n)*(2.0f*(float)TILE_DIM-1.0f)/restime/1000000000.0f);

   nerror=0; 
   for(int i=0; i < n*n; i++)
   {
      if(check_float(c_shadow[i],c_host[i])==0) nerror=nerror+1;
   }
   printf("test comparison shows %d errors\n",nerror);


   hipEventDestroy(start);
   hipEventDestroy(stop);
   hipFree(a_dev);
   hipFree(b_dev);
   hipFree(c_dev);
   hipFree(a_host);
   hipFree(b_host);
   hipFree(c_host);
   hipFree(c_shadow);

}


